#include "hip/hip_runtime.h"

#include <iostream>
#include <math.h>

__global__ void add(int n, float* x, float* y) {
	//int index = threadIdx.x;
	//int stride = blockDim.x;

	//need to take into account new grid of thread blocks
	//index of thread in block
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		y[i] = x[i] * y[i] + x[i];
	}
}

int main(void) {

	int N = 1 << 20;
	//to make these accessible to GPU need to put in Unified memory using cudaMallocManaged()
	//float* x = new float[N];
	//float* y = new float[N];
	float *x, *y;

	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	/* 
	use <<<# blocks, # threads>>> to launch add on GPU, <<<1,1>>> launches one thread to run function on GPU
	will get error in VS on these, but program will still run
	   
	*/
	//calc number of blocks to get N threads
	int blockSize = 1024;
	int numBlocks = (N + blockSize - 1) / blockSize;

	add<<<numBlocks,256>>>(N, x, y);
	//use cudaDeviceSynchronize() to make sure kernel is done before CPU access results
	hipDeviceSynchronize();

	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	}

	std::cout << "Max error: " << maxError << std::endl;
	//to free data in Unified memory need to use cudaFree()
	//delete[] x;
	//delete[] y;

	hipFree(x);
	hipFree(y);

	return 0;
}
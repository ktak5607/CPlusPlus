#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include <stdio.h>
#include <chrono>

using namespace std::chrono;

/*
void addKernel(float* a, float* b, float* c, int N)
{
	for (int i = 0; i < N; i++) {
		c[i] = a[i] * b[i] + a[i];
	}
    
}
*/
const int N = 10000;
__global__ void VecAdd(float* A, float* B, float* C) {
	int i = threadIdx.x;
	int j = threadIdx.y;
	C[i*j] = A[i*j] + B[i*j];
}

int main()
{
	
	
	float* a = new float[N*N];
	float* b = new float[N*N];
	float* c = new float[N*N];
	
	for (int i = 0; i < N*N; i++) {
			a[i] = 1.0f;
			b[i] = 2.0f;
		}
		
	
	
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks = (N / threadsPerBlock.x, N / threadsPerBlock.y);
	VecAdd<< <numBlocks, threadsPerBlock >> > (a, b, c);
	hipDeviceSynchronize();
	/*auto start = high_resolution_clock::now();
	addKernel(a, b, c, N);
	delete(a);
	delete(b);
	delete(c);
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);

	std::cout << duration.count() << std::endl;*/

	return 0;

}
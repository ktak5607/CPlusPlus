#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void mykernel(void) {
	printf("Hellow World\n");
}

__global__ void vecadd_kernel(int* a, int* b, int* c, int N) {
	//int i = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		c[i] = a[i] + b[i];
	}
}
void vecadd(int* a, int* b, int* c, int N) {
	//vecadd_kernel << < 1, N >> > (a, b, c, N);
	vecadd_kernel << < (N+1023)/1024/1024, 1024 >> > (a, b, c, N);
	hipDeviceSynchronize();
	//for (int i = 0; i < N; i++) {
		//vecadd_kernel(a, b, c, N, i);
	//}
}
int main(void) {
	int N = 512;
	int* a, * b, * c;

	hipMallocManaged(&a, N * sizeof(int));
	hipMallocManaged(&b, N * sizeof(int));
	hipMallocManaged(&c, N * sizeof(int));

	//a = (int*)malloc(N * sizeof(int));
	//b = (int*)malloc(N * sizeof(int));
	//c = (int*)malloc(N * sizeof(int));

	vecadd(a, b, c, N);
	//mykernel <<<1, 1 >>> ();
	//hipDeviceSynchronize();

	hipFree(a);
	hipFree(b);
	hipFree(c);
	return 0;
}

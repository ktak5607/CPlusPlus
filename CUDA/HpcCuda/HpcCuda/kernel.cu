#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<iostream>

/* First Exercise
void helloCPU()
{
	std::cout << "Hello from CPU.\n";
}

__global__
void helloGPU()
{
	
	printf("Hello also from GPU.\n");
}

int main()
{

	helloCPU();
	helloGPU<<<5,5>>>();

	return 0;
}
*/

/*Second Exercise
using specific threads and block indicies


__global__ void printSuccessForCorrectExecutionConfiguration()
{
	if (threadIdx.x == 1023 && blockIdx.x == 255)
	{
		printf("Success!\n");
	}
}

int main()
{
	
	printSuccessForCorrectExecutionConfiguration << <256, 1024 >> > ();

	return 0;
}*/

/*Third Exercise
Accelerating a For Loop With a Single Block of Threads
*/
/* FIXME
 * Correct, and refactor 'loop' to be a CUDA Kernel.
 * The new kernel should only do the work
 * of 1 iteration of the original loop.
 */

/*
__global__ void loop()
{
	int i = threadIdx.x;
	printf("This is iteration number %d\n", i);
	
}

int main()
{

	int N = 10;
	loop<<<1,N>>>();
	hipDeviceSynchronize();
}
*/

/*Fourth Excercise
Accelerating a for loop with multiple blocks of threads
*/

/* FIXME
 * Fix and refactor 'loop' to be a CUDA Kernel.
 * The new kernel should only do the work
 * of 1 iteration of the original loop.
 */
/*
#include <cstdio>

__global__ void loop(int N)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < N){
		printf("This is iteration number %d\n", index);
	}
}

int main()
{
	
	int N = 10;
	int numBlcks = 2;
	loop<<<numBlcks, N/numBlcks>>>(N);
	hipDeviceSynchronize();
}
*/

/*Fourth Exercise
array manipulation on both the host and device
*/

//void init(int* a, int N)
//{
//	for (int i = 0; i < N; ++i) {
//		a[i] = i;
//	}
//}
//
///*
// * Double elements in parallel on the GPU.
// */
//
//__global__
//void doubleElements(int* a, int N)
//{
//	int i = blockIdx.x * blockDim.x + threadIdx.x;
//	if (i < N) {
//		a[i] *= 2;
//	}
//}
//
///*
// * Check all elements have been doubled on the host.
// */
//
//bool checkElementsAreDoubled(int* a, int N)
//{
//	for (int i = 0; i < N; ++i) {
//		if (a[i] != i * 2) {
//			return false;
//		}
//	}
//	return true;
//}
//
//int main()
//{
//	int N = 100;
//	int* a;
//
//	size_t size = N * sizeof(int);
//
//	/*
//	 * Refactor this memory allocation to provide a pointer
//	 * 'a' that can be used on both the host and the device.
//	 */
//
//	hipMallocManaged(&a, size);
//
//	init(a, N);
//
//	size_t threads_per_block = 10;
//	size_t number_of_blocks = 10;
//
//	/*
//	 * This launch will not work until the pointer 'a' is also
//	 * available to the device.
//	 */
//
//	doubleElements << <number_of_blocks, threads_per_block >> > (a, N);
//	hipDeviceSynchronize();
//
//	bool areDoubled = checkElementsAreDoubled(a, N);
//	printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");
//
//	/*
//	 * Refactor to free memory that has been allocated to be
//	 * accessed by both the host and the device.
//	 */
//
//	hipFree(a);
//}
/* Fifth Exercise
use a grid-stride loop to manipulate an array larger than the grid
*/
//void init(int* a, int N)
//{
//	for (int i = 0; i < N; ++i) {
//		a[i] = i;
//	}
//}
//
///*
// * In the current application, 'N' is larger than the grid.
// * Refactor this kernel to use a grid-stride loop in order that
// * each parallel thread work on more than one element of the array.
// */
//
//__global__
//void doubleElements(int* a, int N)
//{
//	int indexWithinGrid = blockIdx.x * blockDim.x + threadIdx.x;
//	int stride = gridDim.x * blockDim.x;
//	for(int i = indexWithinGrid; i < N; i+= stride)
//	if (i < N) {
//		a[i] *= 2;
//	}
//}
//
//bool checkElementsAreDoubled(int* a, int N)
//{
//	for (int i = 0; i < N; ++i) {
//		if (a[i] != i * 2) {
//			return false;
//		}
//	}
//	return true;
//}
//
//int main()
//{
//	/*
//	 * 'N' is greater than the size of the grid (see below).
//	 */
//
//	int N = 10000;
//	int* a;
//
//	size_t size = N * sizeof(int);
//	hipMallocManaged(&a, size);
//
//	init(a, N);
//
//	/*
//	 * The size of this grid is 256*32 = 8192.
//	 */
//
//	size_t threads_per_block = 256;
//	size_t number_of_blocks = 32;
//
//	doubleElements << <number_of_blocks, threads_per_block >> > (a, N);
//	hipDeviceSynchronize();
//
//	bool areDoubled = checkElementsAreDoubled(a, N);
//	printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");
//
//	hipFree(a);
//}


/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

void initWith(float num, float* a, int N)
{
	for (int i = 0; i < N; ++i) {
		a[i] = num;
	}
}

/*
 * Device kernel stores into 'result' the sum of each
 * same-indexed value of 'a' and 'b'.
 */

__global__
void addVectorsInto(float* result, float* a, float* b, int N)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < N; i += stride) {
		result[i] = a[i] + b[i];
	}
}

/*
 * Host function to confirm values in 'vector'. This function
 * assumes all values are the same 'target' value.
 */

void checkElementsAre(float target, float* vector, int N)
{
	for (int i = 0; i < N; i++) {
		if (vector[i] != target)
		{
			printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
			exit(1);
		}
	}
	printf("Success! All values calculated correctly.\n");
}

int main()
{
	const int N = 2 << 24;
	size_t size = N * sizeof(float);

	float* a, * b, * c;

	hipMallocManaged(&a, size);
	hipMallocManaged(&b, size);
	hipMallocManaged(&c, size);

	initWith(3, a, N);
	initWith(4, b, N);
	initWith(0, c, N);

	size_t threadsPerBlock;
	size_t numberOfBlocks;

	threadsPerBlock = 80;
	numberOfBlocks = 1;

	addVectorsInto << <numberOfBlocks, threadsPerBlock >> > (c, a, b, N);

	hipDeviceSynchronize();

	checkElementsAre(7, c, N);

	hipFree(a);
	hipFree(b);
	hipFree(c);
	
	return 0;
}